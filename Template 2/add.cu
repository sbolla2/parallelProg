
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add_serial(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

__global__
void add_parallel(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int run_in_parallel = 1;
  int N = 1<<30;
  float *x, *y;
  int blockSize;
  int numBlocks;
  
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU

  // simple serial with 1 thread block

  if( run_in_parallel == 0 )
   add_serial<<<1, 1>>>(N, x, y);

  else{
  // add_parallel code with multiple threads and blocks
  blockSize = 2048;
  numBlocks = (N+ blockSize - 1) / blockSize;
  printf("Invoking add_parallel with numBlocks = %d and blockSize = %d\n", numBlocks, blockSize);
  add_parallel<<<numBlocks, blockSize>>>(N, x, y);
  }
  
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}

